
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>

__global__ auto polynomial_expansion_kernel(const float *const poly,
                                            const int degree, const int n,
                                            float *const array) noexcept
    -> void {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= n) {
    return;
  }

  const auto x = array[idx];
  auto result = 0.0F;
  auto x_to_the_power_of = 1.0F;

  for (auto i = 0; i <= degree; ++i) {
    result += x_to_the_power_of * poly[i];
    x_to_the_power_of *= x;
  }

  array[idx] = result;
}

auto main(const int argc, const char *const *const argv) noexcept -> int {
  if (argc < 3) {
    std::cerr << "usage: " << argv[0] << " n degree\n";
    // return EXIT_FAILURE;
    return -1;
  }

  // atoi is an unsafe function
  const auto n = std::atoi(argv[1]);
  const auto degree = std::atoi(argv[2]);
  const auto num_iter = 1;

  // auto *const array = new float[n];
  // auto *const poly = new float[degree + 1];
  float *array;
  float *poly;
  hipMallocManaged(&array, n * sizeof(float));
  hipMallocManaged(&poly, (degree + 1) * sizeof(float));

  for (auto i = 0; i < n; ++i) {
    array[i] = 1;
  }

  for (auto i = 0; i < degree + 1; ++i) {
    poly[i] = 1;
  }

  const auto begin = std::chrono::system_clock::now();

  // for (auto iter = 0; iter < nbiter; ++iter) {
  //   polynomial_expansion(poly, degree, n, array);
  // }

  // launch GPU kernel
  const auto block_size = 256;
  const auto num_blocks = (n + block_size - 1) / block_size;
  polynomial_expansion_kernel<<<num_blocks, block_size>>>(poly, degree, n,
                                                          array);
  // wait for GPU to finish
  hipDeviceSynchronize();

  const auto end = std::chrono::system_clock::now();
  const auto total_time =
      std::chrono::duration<double>{(end - begin) / num_iter};

  // check results
  auto correct = true;
  int ind;
  for (auto i = 0; i < n; ++i) {
    if (std::fabs(array[i] - (degree + 1)) > 0.01) {
      correct = false;
      ind = i;
    }
  }

  if (!correct) {
    std::cerr << "Result is incorrect. In particular array[" << ind
              << "] should be " << degree + 1 << " not " << array[ind] << '\n';
  }

  std::cerr << array[0] << '\n';
  std::cout << n << " " << degree << " " << total_time.count() << '\n';

  // clean up
  // delete[] array;
  // delete[] poly;
  hipFree(array);
  hipFree(poly);

  return EXIT_SUCCESS;
}
